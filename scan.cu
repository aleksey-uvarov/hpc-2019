#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
// #include <stdexcept>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


__global__ void prepare_function(float * d_out, int n_points,
                                 float x_min, float x_max)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float x = x_min + (x_max - x_min) * id / n_points;
    d_out[id] = exp(-pow(x, 2));
//     d_out[id] = (float) id;
}


// __global__ void blelloch_reduce(float * d_in, int n_points)
// {
//     /* Assuming n_points is a power of two */
//     int n_current = 2;
//     int id = threadIdx.x + blockIdx.x * blockDim.x;
//     
//     while(n_current <= n_points)
//     {
//         if ((id + 1) % n_current == 0)
//         {
//             d_in[id] += d_in[id - n_current/2];
//         }
//         __syncthreads();
//         n_current = n_current * 2;
//     }
//     
// }


__global__ void blelloch_reduce_step(float * d_in, int n_points, int stride_step)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if ((id + 1) % stride_step == 0)
    {
        d_in[id] += d_in[id - stride_step/2];
    }
}


__global__ void kill_last(float * d_in, int n_points)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;       
   
    if (id == n_points - 1)
    {
        d_in[id] = 0;
    }
}


/*
__global__ void blelloch_downsweep(float * d_in, int n_points)
{
    int n_current = n_points;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float tmp;
    
    if (id == n_points - 1)
    {
        d_in[id] = 0;
    }
    __syncthreads();
    
    while(n_current >= 2)
    {
        if ((id + 1) % n_current == 0)
        {
            tmp = d_in[id];
            d_in[id] += d_in[id - n_current/2];
            d_in[id - n_current/2] = tmp;
        }
        n_current = n_current / 2;
        __syncthreads();
    }
}*/



__global__ void blelloch_downsweep_step(float * d_in, int n_points, int stride_step)
{
    
    int id = threadIdx.x + blockIdx.x * blockDim.x;       
    float tmp;
  
 
    if ((id + 1) % stride_step == 0)
    {
        tmp = d_in[id];
        d_in[id] += d_in[id - stride_step/2];
        d_in[id - stride_step/2] = tmp;
    }
//     __syncthreads();
}


int main(int argc, char* argv[])
{
    float minus_infty = -8;
    float x_max = 0;
    int n_blocks = 1024;
    int n_points_per_block = 1024;
    int n_points = n_points_per_block * n_blocks;
    int stride_step;
    
    float dx;
    float *devFunVals;
    float *hostFunVals;
    float *hostFunVals2;
    
    if (argc > 1)
    {
        sscanf(argv[1], "%f", &x_max);
//         printf("%f\n", x_max);
        if (x_max < minus_infty)
        {
            printf("0\n");
            return 0;
        }
    }
    else
    {
        printf("Usage: ./scan <number> \n");
        return 0;
    }
    dx = (x_max - minus_infty) / (float) n_points;
//     printf("dx: %e\n", dx);
//     printf("n_points: %d\n", n_points);
//     
    if (n_points < 0 || ((n_points & (n_points - 1)) != 0))
    {
        printf("n_points is not a power of two\n");
        return 1;
    }
    
    hostFunVals = (float *)calloc(n_points, sizeof(float));
    hostFunVals2 = (float *)calloc(n_points, sizeof(float));
    
    CUDA_CALL(hipMalloc((void **)&devFunVals, n_points*sizeof(float)));

    
    prepare_function<<<n_blocks, n_points_per_block>>>(devFunVals, n_points, minus_infty, x_max);
    
//     blelloch_reduce<<<n_blocks, n_points_per_block>>>(devFunVals, n_points);

    stride_step = 2;
    while(stride_step <= n_points)
    {
        blelloch_reduce_step<<<n_blocks, n_points_per_block>>>(devFunVals, n_points, stride_step);        
        
        stride_step = stride_step * 2;
    }
    stride_step = n_points;

    
    CUDA_CALL(hipMemcpy(hostFunVals, devFunVals, n_points*sizeof(float), hipMemcpyDeviceToHost));
    
    kill_last<<<n_blocks, n_points_per_block>>>(devFunVals, n_points);
    
    while(stride_step >= 2)
    {
        blelloch_downsweep_step<<<n_blocks, n_points_per_block>>>(devFunVals, n_points, stride_step);
        stride_step = stride_step / 2; 
    }


    CUDA_CALL(hipMemcpy(hostFunVals2, devFunVals, n_points*sizeof(float), hipMemcpyDeviceToHost));

//     for(int i=0; i<n_points; i++)
//     {
//         printf("%1.4f \n", hostFunVals[i]);
//     }
//     printf("=========\n");
//     
//     for(int i=0; i<n_points; i++)
//     {
//         printf("%1.4f \n", hostFunVals2[i]);
//     }
//     printf("\n");
//     printf("Func value: %1.5f\n", hostFunVals[n_points - 1]);
    printf("Integral value: %1.5e\n", hostFunVals2[n_points - 1] * dx);
    
    
    
    
    
    
    
    
    
    
    
    
    free(hostFunVals);
    free(hostFunVals2);

    CUDA_CALL(hipFree(devFunVals));


    
    
    return 0;    
}
